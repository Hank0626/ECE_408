#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define MASK_RADIUS 1
#define TILE_SIZE 8
//@@ Define constant memory for device kernel here
__constant__ float M[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  __shared__ float N_ds[TILE_SIZE + MASK_WIDTH -1][TILE_SIZE + MASK_WIDTH -1][TILE_SIZE + MASK_WIDTH -1];

  int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z;
  int x_o = blockIdx.x * TILE_SIZE + tx;
  int y_o = blockIdx.y * TILE_SIZE + ty;
  int z_o = blockIdx.z * TILE_SIZE + tz;
  int x_i = x_o - MASK_RADIUS;
  int y_i = y_o - MASK_RADIUS;
  int z_i = z_o - MASK_RADIUS;
  
  if ((x_i >= 0) && (x_i < x_size) && (y_i >= 0) && (y_i < y_size) && (z_i >= 0) && (z_i < z_size)) {
    N_ds[tz][ty][tx] = input[z_i * (y_size * x_size) + y_i * x_size + x_i];
  }
  else N_ds[tz][ty][tx] = 0;

  __syncthreads();

  float out = 0.0;
  if (tx < TILE_SIZE && ty < TILE_SIZE && tz < TILE_SIZE) {
    for (int i = 0; i < MASK_WIDTH; i++) {
      for (int j = 0; j < MASK_WIDTH; j++) {
        for (int k = 0; k < MASK_WIDTH; k++) {
          out += M[i][j][k] * N_ds[i + tz][j + ty][k + tx];
        }
      }
    }
    if (x_o < x_size && y_o < y_size && z_o < z_size) {
      output[z_o * (y_size * x_size) + y_o * x_size + x_o] = out;
    }
  }
}


int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void**) &deviceInput, (inputLength - 3) * sizeof(float));
  hipMalloc((void**) &deviceOutput, (inputLength - 3) * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, &hostInput[3], (inputLength - 3) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M), hostKernel, kernelLength * sizeof(float));
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimgrid(ceil(x_size / (1.0 * TILE_SIZE)), ceil(y_size / (1.0 * TILE_SIZE)), ceil(z_size / (1.0 * TILE_SIZE)));
  dim3 dimblock(TILE_SIZE + MASK_WIDTH - 1, TILE_SIZE + MASK_WIDTH - 1, TILE_SIZE + MASK_WIDTH - 1);
  //@@ Launch the GPU kernel here
  hipDeviceSynchronize();
  conv3d<<<dimgrid, dimblock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(&hostOutput[3], deviceOutput, (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
